
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

__global__ void scan(float *d_in,float *d_out,const int size){
	int idx = threadIdx.x;
	d_out[idx] = d_in[idx];
	__syncthreads();
	float out;
	for(int step=1;step<size;step*=2){
		if(idx-step>=0){
			out = d_out[idx]+d_out[idx-step];
			/*
			__syncthreads();
			d_out[idx] = out;
			__syncthreads();
			*/
		}
		
		__syncthreads();
		if(idx-step>0){
			d_out[idx] = out;
		}
		__syncthreads();

	}
}

void init(float *h_in,const int size){
	for(int i=0;i<size;i++)
		h_in[i] = i;
}

int main(){
	int size = 1024;
	float *h_in,*h_out;
	float *d_in,*d_out;
	h_in = (float *)malloc(size*sizeof(float));
	h_out = (float *)malloc(size*sizeof(float));
	init(h_in,size);
	printf("array:");
	for(int i=0;i<size;i++){
		printf("%f ",h_in[i]);
	}
	printf("\n");
	hipMalloc((float **)&d_in,size*sizeof(float));
	hipMalloc((float **)&d_out,size*sizeof(float));
	hipMemcpy(d_in,h_in,size*sizeof(float),hipMemcpyHostToDevice);
	time_t t_start = clock();
	scan<<<1,size>>>(d_in,d_out,size);
	time_t t_end = clock();
	hipMemcpy(h_out,d_out,size*sizeof(float),hipMemcpyDeviceToHost);
	printf("time:%fms\n",difftime(t_end,t_start));
	printf("result:");
	for(int i=0;i<size;i++){
		printf("%f ",h_out[i]);
	}
	printf("\n");
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}

