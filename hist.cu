
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>

void init_in(int *h_in,const int size){
	srand((unsigned int)time(NULL));
	for(int i=0;i<size;i++)
		h_in[i] = rand()%size;
}

void init_out(int *h_out,const int size){
	for(int i=0;i<size;i++)
		h_out[i] = 0;
}

void hist_normal(int *h_in,int *h_out,const int size,const int bin_size){
	for(int i=0;i<size;i++){
		int no = h_in[i]/bin_size;
		h_out[no]++;
	}
}

__global__ void hist_atomic(int *d_in,int *d_out,const int bin_size){
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	int no = d_in[idx]/bin_size;
	atomicAdd(&d_out[no],1);
}

__global__ void hist_local(int *d_in,int *d_out,const int bin_size,const int thread_size){
	int idx = threadIdx.x;
	int l_out[32] = {0};
	for(int i=idx*thread_size;i<idx*thread_size+thread_size;i++){
		int no = d_in[i]/bin_size;
		l_out[no]++;
	}
	for(int i=0;i<32;i++)
		atomicAdd(&d_out[i],l_out[i]);
}

void show_hist(int *h_out,const int bin_num){
	for(int i=0;i<bin_num;i++)
		printf("%d : %d\n",i+1,h_out[i]);
	printf("\n");
}

int main(){
	int size = 262144;
	int bin_num = 32;
	int bin_size = size/bin_num;
	int *h_in,*h_out;
	int *d_in,*d_out;
	h_in = (int *)malloc(size*sizeof(int));
	h_out = (int *)malloc(bin_num*sizeof(int));
	init_in(h_in,size);
	init_out(h_out,bin_num);
	time_t t_start = clock();
	hist_normal(h_in,h_out,size,bin_size);
	time_t t_end = clock();
	printf("hist_normal_time: %fms\n",difftime(t_end,t_start));
	printf("hist_normal_result: \n");
	show_hist(h_out,bin_num);

	hipMalloc((int **)&d_in,size*sizeof(int));
	hipMalloc((int **)&d_out,bin_num*sizeof(int));
	hipMemcpy(d_in,h_in,size*sizeof(int),hipMemcpyHostToDevice);
	dim3 block1(256);
	dim3 thread1(1024);
	t_start = clock();
	hist_atomic<<<block1,thread1>>>(d_in,d_out,bin_size);
	t_end = clock();
	hipMemcpy(h_out,d_out,bin_num*sizeof(int),hipMemcpyDeviceToHost);
	printf("hist_atomic_time: %fms\n",difftime(t_end,t_start));
	printf("hist_atomic_result: \n");
	show_hist(h_out,bin_num);

	int n = 64;
	dim3 thread2(n);
	init_out(h_out,bin_num);
	hipMemcpy(d_out,h_out,bin_num*sizeof(int),hipMemcpyHostToDevice);
	t_start = clock();
	hist_local<<<1,thread2>>>(d_in,d_out,bin_size,size/n);
	t_end =clock();
	hipMemcpy(h_out,d_out,bin_num*sizeof(int),hipMemcpyDeviceToHost);
	printf("hist_local_time: %fms\n",difftime(t_end,t_start));
	printf("hist_local_result: \n");
	show_hist(h_out,bin_num);

	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);

	hipDeviceReset();
	return 0;
}


