
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
//全局内存
__global__ void global_reduce(float *d_in,float *d_out){
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idxn = threadIdx.x;
	for(int s = blockDim.x/2;s>0;s>>=1){
		if(idxn<s){
			d_in[idx] += d_in[idx+s];
		}
		__syncthreads();//同步
	}
	if(idxn == 0){
		d_out[blockIdx.x] = d_in[idx];
	}
}
//共享内存
__global__ void shared_reduce(float *d_in,float *d_out){
	extern __shared__ float s_in[];
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int idxn = threadIdx.x;
	s_in[idxn] = d_in[idx];
	__syncthreads();
	for(int s = blockDim.x/2;s>0;s>>=1){
		if(idxn<s){
			s_in[idxn] += s_in[idxn+s];
		}
		__syncthreads();//同步
	}
	if(idxn == 0){
		d_out[blockIdx.x] = s_in[0];
	}
}

void init(float *h_in,const int size){
	srand((unsigned int)time(NULL));
	for(int i=0;i<size;i++)
		h_in[i] =(float)(rand()%101)/100.0f;
}

int main(){
	int size = 1024;
	float *h_in;
	float h_out = 0;
	h_in = (float *)malloc(size*size*sizeof(float));
	init(h_in,size*size);//初始化
	time_t t_start = clock();
	for(int i=0;i<size*size;i++){
		h_out += h_in[i];
	}
	time_t t_end = clock();
	printf("CPU sum:%f\n",h_out);
	printf("CPU time:%fms\n",difftime(t_end,t_start));
	float *d_in;
	float *d_out;
	float *d_out_mid;
	dim3 block(size);
	dim3 thread(size);
	hipMalloc((float **)&d_in,size*size*sizeof(float));
	hipMalloc((float **)&d_out_mid,size*sizeof(float));
	hipMalloc((float **)&d_out,sizeof(float));
	hipMemcpy(d_in,h_in,size*size*sizeof(float),hipMemcpyHostToDevice);
	t_start = clock();
	global_reduce<<<block,thread>>>(d_in,d_out_mid);
	global_reduce<<<1,thread>>>(d_out_mid,d_out);
	t_end = clock();
	hipMemcpy(&h_out,d_out,sizeof(float),hipMemcpyDeviceToHost);
	printf("GPU(global) sum:%f\n",h_out);
	printf("GPU(global) time:%fms\n",difftime(t_end,t_start));

	hipMemcpy(d_in,h_in,size*size*sizeof(float),hipMemcpyHostToDevice);
	t_start = clock();
	shared_reduce<<<block,thread,size*sizeof(float)>>>(d_in,d_out_mid);
	shared_reduce<<<1,thread,size*sizeof(float)>>>(d_out_mid,d_out);
	t_end = clock();
	hipMemcpy(&h_out,d_out,sizeof(float),hipMemcpyDeviceToHost);
	printf("GPU(shared) sum:%f\n",h_out);
	printf("GPU(shared) time:%fms\n",difftime(t_end,t_start));

	free(h_in);
	hipFree(d_in);
	hipFree(d_out_mid);
	hipFree(d_out);
	hipDeviceReset();//重置当前资源
	return 0;
}
