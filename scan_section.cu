
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
using namespace std;
__global__ void mul(int *d_in1,int *d_in2,int *d_out){
	int idx = threadIdx.x;
	d_out[idx] = d_in1[idx]*d_in2[idx];
}
__global__ void reduce_section(int *d_in,int &d_out,const int start,const int end){
	int idx = threadIdx.x;
	extern __shared__ int s_out[];
	s_out[idx] = d_in[start+idx];
	__syncthreads();
	int out;
	for(int step=1;step<end-start;step*=2){
		if(idx-step>=0){
			out = s_out[idx]+s_out[idx-1];
		}
		__syncthreads();
		if(idx-step>=0)
			s_out[idx] = out;
		__syncthreads();
	}
	if(idx == end-start-1)
		d_out = s_out[idx];
}
int main(){
	const int size = 6;
	int value[size] = {1,2,3,4,5,6};
	int cols[size] = {0,2,1,0,1,0};
	int rows[5] = {0,2,3,5,size};//最后一个元素记录非零元素个数
	int mul_val[3] = {1,2,3};
	int mul_valn[size];//非零元素相乘的对应元素
	printf("左矩阵:\n");
	int flag = 0;
	for(int i=0;i<4;i++){
		for(int i=0;i<3;i++){
			if(i == cols[flag])
				printf("%d ",value[flag++]);
			else
				printf("0 ");
		}
		printf("\n");
	}
	printf("\n右矩阵:\n");
	for(int i=0;i<3;i++){
		printf("%d\n",mul_val[i]);
	}
	printf("\n");
	for(int i=0;i<size;i++){
		mul_valn[i] = mul_val[cols[i]];
	}
	int *h_in1 = value;
	int *h_in2 = mul_valn;
	int *h_out;
	int *d_in1;
	int *d_in2;
	int *d_out_mid;
	int *d_out;
	h_out = (int *)malloc(4*sizeof(int));
	hipMalloc((int **)&d_in1,size*sizeof(int));
	hipMalloc((int **)&d_in2,size*sizeof(int));
	hipMalloc((int **)&d_out,4*sizeof(int));
	hipMalloc((int **)&d_out_mid,size*sizeof(int));
	hipMemcpy(d_in1,h_in1,size*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_in2,h_in2,size*sizeof(int),hipMemcpyHostToDevice);
	dim3 thread(size);
	mul<<<1,thread>>>(d_in1,d_in2,d_out_mid);
	for(int i=1;i<5;i++){
		int sizenew = rows[i]-rows[i-1];
		dim3 threadnew(sizenew);
		reduce_section<<<1,threadnew,sizenew>>>(d_out_mid,d_out[i-1],rows[i-1],rows[i]);
	}
	hipMemcpy(h_out,d_out,4*sizeof(int),hipMemcpyDeviceToHost);
	printf("结果:\n");
	for(int i=0;i<4;i++){
		printf("%d\n",h_out[i]);
	}
	printf("\n");
	free(h_out);
	hipFree(d_in1);
	hipFree(d_in2);
	hipFree(d_out_mid);
	hipFree(d_out);
	return 0;
}
