
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<unistd.h>

__global__ void sumArraysOnGPUN(float *A,float *B,float *C,const int N){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if(idx<N)
		C[idx] = A[idx] + B[idx];
	printf(" %f + %f = %f   On GPU:block %d thread %d\n",A[idx],B[idx],C[idx],blockIdx.x,threadIdx.x);
}
void initialData(float *ip,const int size){
	time_t t;
	srand((unsigned int)time(&t));
	for(int i=0;i<size;i++){
		ip[i] = (float)(rand()%100)/1.0f;
	}
}
void print(float *array,const int size){
	for(int i=0;i<size;i++){
		printf(" %f",array[i]);
	}
	printf("\n");
}

int main(){
	int n;
	scanf("%d",&n);
	int nBytes = n*sizeof(float);
	float *h_A,*h_B,*h_C;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	h_C = (float *)malloc(nBytes);
	initialData(h_A,n);
	sleep(1);
	initialData(h_B,n);
	print(h_A,n);
	print(h_B,n);
	float *d_A,*d_B,*d_C;
	hipMalloc((float **)&d_A,nBytes);
	hipMalloc((float **)&d_B,nBytes);
	hipMalloc((float **)&d_C,nBytes);
	hipMemcpy(d_A,h_A,nBytes,hipMemcpyHostToDevice);
	hipMemcpy(d_B,h_B,nBytes,hipMemcpyHostToDevice);
	dim3 block(1);
	dim3 thread(n);
	sumArraysOnGPUN<<<block,thread>>>(d_A,d_B,d_C,n);
	hipMemcpy(h_C,d_C,nBytes,hipMemcpyDeviceToHost);
	print(h_C,n);
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}
