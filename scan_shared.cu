
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
__global__ void scan_shared(float *d_in,float *d_out,const int size){
	extern __shared__ float s_in[];
	int idx = threadIdx.x;
	s_in[idx] = d_in[idx];
	__syncthreads();
	float out;
	for(int step=1;step<size;step*=2){
		if(idx-step>=0){
			out = s_in[idx]+s_in[idx-step];
		}
		__syncthreads();
		if(idx-step>=0){
			s_in[idx] = out;
		}
		__syncthreads();
	}
	d_out[idx] = s_in[idx];
}

void init(float *p,const int size){
	for(int i=0;i<size;i++)
		p[i] = i;
}

int main(){
	int size = 1024;
	float *h_in,*h_out;
	float *d_in,*d_out;
	h_in = (float *)malloc(size*sizeof(float));
	h_out = (float *)malloc(size*sizeof(float));
	init(h_in,size);
	printf("array:");
	for(int i=0;i<size;i++)
		printf("%f ",h_in[i]);
	printf("\n");
	hipMalloc((float **)&d_in,size*sizeof(float));
	hipMalloc((float **)&d_out,size*sizeof(float));
	hipMemcpy(d_in,h_in,size*sizeof(float),hipMemcpyHostToDevice);
	time_t t_start = clock();
	scan_shared<<<1,size,size*sizeof(float)>>>(d_in,d_out,size);
	time_t t_end = clock();
	hipMemcpy(h_out,d_out,size*sizeof(float),hipMemcpyDeviceToHost);
	printf("time:%fms\n",difftime(t_end,t_start));
	printf("result:");
	for(int i=0;i<size;i++)
		printf("%f ",h_out[i]);
	printf("\n");
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}
