
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
__global__ void oddevenSort(int *d_in,int size,int oe_flag,int &d_ch_flag){
	int idx = threadIdx.x+blockIdx.x*blockDim.x;
	int p = 2*idx+oe_flag;
	if(p+1<size){
		if(d_in[p]>d_in[p+1]){
			int temp = d_in[p];
			d_in[p] = d_in[p+1];
			d_in[p+1] = temp;
			d_ch_flag = 1;
		}
	}
}
void init(int *p,const int size){
	srand((unsigned int)time(NULL));
	for(int i=0;i<size;i++){
		p[i] = rand()%size;
	}
}
void show(int *p,const int size){
	for(int i=0;i<size;i++){
		printf("%d ",p[i]);
	}
	printf("\n");
}
void bubbleSort(int *p,const int size){
	for(int i=0;i<size-1;i++){
		for(int j=0;j<size-i-1;j++){
			if(p[j]>p[j+1]){
				int temp = p[j];
				p[j] = p[j+1];
				p[j+1] = temp;
			}
		}
	}
}
int main(){
	int size = 10*1024;
	int *h_in;
	int *h_out;
	h_in = (int *)malloc(size*sizeof(int));
	h_out = (int *)malloc(size*sizeof(int));
	init(h_in,size);
	//show(h_in,size);
	//printf("\n");
	int *d_in;
	int *d_ch_flag;
	hipMalloc((int **)&d_in,size*sizeof(int));
	hipMalloc((int **)&d_ch_flag,sizeof(int));
	hipMemcpy(d_in,h_in,size*sizeof(int),hipMemcpyHostToDevice);
	int oe_flag = 0;//判断当前进行排序的类型
	int ch_flag = 1;//判断数组是否发生改变
	dim3 block(10);
	dim3 thread(1024/2);
	time_t t_start = clock();
	while(ch_flag||oe_flag){//偶排序和奇排序必须成对出现
		ch_flag = 0;
		hipMemcpy(d_ch_flag,&ch_flag,sizeof(int),hipMemcpyHostToDevice);
		oddevenSort<<<block,thread>>>(d_in,size,oe_flag,d_ch_flag[0]);
		hipMemcpy(&ch_flag,d_ch_flag,sizeof(int),hipMemcpyDeviceToHost);
		oe_flag = 1^oe_flag;
	}
	time_t t_end = clock();
	hipMemcpy(h_out,d_in,size*sizeof(int),hipMemcpyDeviceToHost);
	//show(h_out,size);
	printf("GPU time:%fms\n",difftime(t_end,t_start));
	t_start = clock();
	bubbleSort(h_in,size);
	t_end = clock();
	printf("CPU time:%fms\n",difftime(t_end,t_start));
	free(h_in);
	free(h_out);
	hipFree(d_in);
	hipFree(d_ch_flag);
	return 0;
}
